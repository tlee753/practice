
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <time.h>

using namespace std;

__global__
void sumation (int max, int *sumPtr) {
	printf("%d\n", max);
	for (int i = 1; i <= max; i++) {
		*sumPtr += i;
		printf("%d\n", *sumPtr);
	}
}

int main() {
	clock_t t1, t2;
	t1 = clock();
	
	cout << "Program started..." << endl;
	int x = 800, sum = 0;
	int *sumPtr = &sum;
	
	hipMallocManaged(&sumPtr, sizeof(int));
	
	// cout << "Pick a number to sum: " << endl;
	// cin >> x;
	
	sumation<<<1, 256>>>(x, sumPtr);
	hipDeviceSynchronize();
	
	cout << "Sum is " << sum << "." << endl;
	
	t2 = clock();
	float diff = (float)t2 - (float)t1;
	float seconds = diff/ CLOCKS_PER_SEC;
	cout << "Program time is " << seconds << "." << endl;
	
	hipFree(sumPtr);
	
	return 0;
}
